#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "ftocmacros.h"
#include "F2Cinclude.h"
#define TRUE 1
#define FALSE 0
#define IMS 1
#define IME 50
#define KMS 1
#define KME 20
   __shared__ float t3dS[100];
typedef struct {
  float rd;
  int kms,kme,ims,ime;
} blockshared_s1;
//!ACC$REGION(<ime-ims+1:block=2>,<kme-kms+1>,<kms,kme,ims,ime,rd:in>,<t3d:inout>,<t3d:inout,shared,demote(2)>) BEGIN
__global__ void blockshared_Kernel1(blockshared_s1 blockshared_k1,float *t3d) {
  int kms = blockshared_k1.kms;
  int kme = blockshared_k1.kme;
  int ims = blockshared_k1.ims;
  int ime = blockshared_k1.ime;
  float rd = blockshared_k1.rd;
// Generated copy from GPU global to shared memory
  int iS1,iS2,iS3,iS4,iS5,iS6,iS7;
  t3dS[FTNREF1D(threadIdx.x,0)] = t3d[FTNREF2D(threadIdx.x%(ime-ims+1)+ims,(blockIdx.x*2)+(threadIdx.x/(blockDim.x/2))+kms,ime-ims+1,ims,kms)];

  int k;
  int i;
//! standard loop test with simple array indices
//!ACC$DO PARALLEL(1)
  k = (blockIdx.x*2)+(threadIdx.x/(blockDim.x/2))+kms;
//  for (k=kms;k<=kme;k++) {
//!ACC$DO VECTOR(1)
    i = threadIdx.x%(blockDim.x/2)+ims;
//    for (i=ims;i<=ime;i++) {
      t3dS[FTNREF1D(threadIdx.x,0)] = t3dS[FTNREF1D(threadIdx.x,0)] * rd;
//    }
//! test to handle multiple terms in the affected index "i"
//! note how the base thread is 2, so data access by each thread is consistent
//!ACC$DO VECTOR(1,1:ime-1)
    i = threadIdx.x%(blockDim.x/2)+1;
    if ((i >= 1) && (i <= ime-1)) {
//    for (i=ims;i<=ime - 1;i++) {
      t3dS[FTNREF1D(threadIdx.x,0)] = t3dS[FTNREF1D(threadIdx.x,0)] * rd;
//    }
    }
//!ACC$THREAD(ime-1)
if (threadIdx.x%(ime-ims+1) == ime-1) {
    t3dS[FTNREF1D((threadIdx.x/(ime-ims+1))*(ime-ims+1)+ime-1,0)] = 0.F;
}
//  }
// Generated copy from GPU shared to global memory
  t3d[FTNREF2D(threadIdx.x%(ime-ims+1)+ims,(blockIdx.x*2)+(threadIdx.x/(blockDim.x/2))+kms,ime-ims+1,ims,kms)] = t3dS[FTNREF1D(threadIdx.x,0)];

  return;
}
//!ACC$REGION END
//! Unit test to demonstrate threading over multiple array dimensions.
//!
//! Designed for GPU translation
//! Mark Govett
//!
extern "C" void blockshared_ (float *rd__G,float *t3d,int *ims__G,int *ime__G,int *kms__G,int *kme__G) {

  static float runtime = 0.0;
  float elapsed_time_ms = 0.f;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float rd=*rd__G;
  int ims=*ims__G;
  int ime=*ime__G;
  int kms=*kms__G;
  int kme=*kme__G;
hipError_t hipError_t;
  float *D_t3d;
  hipError_t = hipMalloc((void **) &D_t3d,((ime-ims+1)*(kme-kms+1))*sizeof(float));
  cudaErrorChk("blockShared.f90.ACC.1",13,"t3d",hipError_t);
//!ACC$DATA(<t3d:none,shared(100)>)


  dim3 cuda_threads1((ime-ims+1)*2);
  dim3 cuda_grids1((kme-kms+1)/2);

  unsigned int timer = 0;

  blockshared_s1 blockshared_k1;
  blockshared_k1.kms = kms;
  blockshared_k1.kme = kme;
  blockshared_k1.ims = ims;
  blockshared_k1.ime = ime;
  blockshared_k1.rd = rd;
  hipError_t = hipMemcpy(D_t3d,t3d,((ime-ims+1)*(kme-kms+1))*sizeof(float),hipMemcpyHostToDevice);
  cudaErrorChk("blockShared.f90.ACC.1",13,"t3d",hipError_t);

  hipEventRecord( start, 0 );
  hipEventSynchronize(start);
  // setup execution parameters 
  // execute the kernel 
  blockshared_Kernel1<<< cuda_grids1, cuda_threads1 >>>(blockshared_k1,D_t3d);
  hipDeviceSynchronize();

  hipEventRecord( stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime( &elapsed_time_ms, start, stop );
  runtime += elapsed_time_ms;
  //  printf("blockshared kernel execution time: %.2f ms\n",runtime);
  hipError_t = hipMemcpy(t3d,D_t3d,((ime-ims+1)*(kme-kms+1))*sizeof(float),hipMemcpyDeviceToHost);
  cudaErrorChk("blockShared.f90.ACC.1",30,"t3d",hipError_t);
  hipError_t = hipFree(D_t3d);
  cudaErrorChk("blockShared.f90.ACC.1",31,"t3d",hipError_t);

  return;
}
