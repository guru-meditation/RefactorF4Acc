#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "ftocmacros.h"
#include "F2Cinclude.h"
#define TRUE 1
#define FALSE 0
typedef struct {
  int nz,nip;
} accdata_s1;
typedef struct {
  float factor;
  int nz,nip;
} accdata_s2;
//!ACC$REGION(<nz>,<nip>,<flx:in>,<nz,nip:in>,<vol:none,extern>) BEGIN
__global__ void accdata_Kernel1(float *flx,accdata_s1 accdata_k1,float *vol) {
  int nz = accdata_k1.nz;
  int nip = accdata_k1.nip;

  int ipn;
  int k;
//!acc$do parallel
  ipn = blockIdx.x+1;
//  for (ipn=1;ipn<=nip;ipn++) {
//!acc$do vector
    k = threadIdx.x+1;
//    for (k=1;k<=nz;k++) {
      flx[FTNREF2D(k,ipn,nz,1,1)] = flx[FTNREF2D(k,ipn,nz,1,1)] / vol[FTNREF2D(k,ipn,nz,1,1)];
//    }
//  }

  return;
}
//!ACC$REGION END
//!ACC$REGION(<nz>,<nip>,<flx:out>,<nz,nip:in>,<vol:none,extern>,<factor:in>) BEGIN
__global__ void accdata_Kernel2(float *flx,accdata_s2 accdata_k2,float *vol) {
  int nz = accdata_k2.nz;
  int nip = accdata_k2.nip;
  float factor = accdata_k2.factor;

  int ipn;
  int k;
//!acc$do parallel
  ipn = blockIdx.x+1;
//  for (ipn=1;ipn<=nip;ipn++) {
//!acc$do vector
    k = threadIdx.x+1;
//    for (k=1;k<=nz;k++) {
      flx[FTNREF2D(k,ipn,nz,1,1)] = flx[FTNREF2D(k,ipn,nz,1,1)] / (factor * vol[FTNREF2D(k,ipn,nz,1,1)]);
//    }
//  }

  return;
}
//!ACC$REGION END
//! Unit tests for DATA_MOVEMENT
//!
//! Designed for GPU translation and execution
//! Mark Govett       Nov 2010
//! NOAA/ESRL
//! ACC$REGION
//!   Variables must be defined either as input arguments, or as constants. 
//!
extern "C" void accdata_ (float *vol,float *flx,float *factor__G) {

  static float runtime = 0.0;
  float elapsed_time_ms = 0.f;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float factor=*factor__G;
int nz=5;

int nip=10;
hipError_t hipError_t;
  float *D_flx;
  hipError_t = hipMalloc((void **) &D_flx,((nz)*(nip))*sizeof(float));
  cudaErrorChk("accdata.f90.ACC.1",37,"flx",hipError_t);
  extern float *D_vol;
//! GPU constants scope is restricted to all routines within a single file. A
//! namespace conflict will occur If multiple routines declare the same constant 
//! variable.  Since the fermi architecture which supports cache, there appears to
//! be little benefit to using constant memory.  Constant variable are always 
//! define with intent "in"; anything else is an error
//!! MWG: error noted in F2C-ACC_V4.8 
//!!ACC$DATA(<factor:in,constant>)
//! All variables used in the GPU region must be listed as arguments in the 
//! accelerated region (ACC$REGION directive). The F2C-ACC option --Analysis=1
//! will analyze each region and determine a recommended (and conservative) list
//! of intents for each variable in the region.
//!
//! In this region, nz and nip are scalars that are copied to the GPU.
//!
//! vol was copied to the GPU in copyIn and defined with global scope.  Therefore
//! it an be referenced as an externally defined GPU resident variable.
//!
//! flux was defined with "in" scope so it will be copied to gpu prior to
//! execution of hte kernel.  It will remain GPU resident until this routine
//! is complete.  

  dim3 cuda_threads1(nz);
  dim3 cuda_grids1(nip);

  unsigned int timer = 0;

  hipError_t = hipMemcpy(D_flx,flx,((nz)*(nip))*sizeof(float),hipMemcpyHostToDevice);
  cudaErrorChk("accdata.f90.ACC.1",37,"flx",hipError_t);
  accdata_s1 accdata_k1;
  accdata_k1.nz = nz;
  accdata_k1.nip = nip;

  hipEventRecord( start, 0 );
  hipEventSynchronize(start);
  // setup execution parameters 
  // execute the kernel 
  accdata_Kernel1<<< cuda_grids1, cuda_threads1 >>>(D_flx,accdata_k1,D_vol);
  hipDeviceSynchronize();

  hipEventRecord( stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime( &elapsed_time_ms, start, stop );
  runtime += elapsed_time_ms;
  //  printf("accdata kernel execution time: %.2f ms\n",runtime);
//!flx is defined with intent out so it is copied to the CPU when the GPU region
//! has completed.  Based on the intent of the previous region (intent "in"), the
//! variable is resident on the GPU.

  dim3 cuda_threads2(nz);
  dim3 cuda_grids2(nip);


  accdata_s2 accdata_k2;
  accdata_k2.nz = nz;
  accdata_k2.nip = nip;
  accdata_k2.factor = factor;
  accdata_k2.factor = factor;

  hipEventRecord( start, 0 );
  hipEventSynchronize(start);
  // setup execution parameters 
  // execute the kernel 
  accdata_Kernel2<<< cuda_grids2, cuda_threads2 >>>(D_flx,accdata_k2,D_vol);
  hipDeviceSynchronize();

  hipEventRecord( stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime( &elapsed_time_ms, start, stop );
  runtime += elapsed_time_ms;
  //  printf("accdata kernel execution time: %.2f ms\n",runtime);
  hipError_t = hipMemcpy(flx,D_flx,((nz)*(nip))*sizeof(float),hipMemcpyDeviceToHost);
  cudaErrorChk("accdata.f90.ACC.1",57,"flx",hipError_t);
  hipError_t = hipFree(D_flx);
  cudaErrorChk("accdata.f90.ACC.1",58,"flx",hipError_t);

}
