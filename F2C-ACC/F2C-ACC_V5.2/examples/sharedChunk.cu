#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "ftocmacros.h"
#include "F2Cinclude.h"
#define TRUE 1
#define FALSE 0
   __shared__ int operandS[50*50];
typedef struct {
  int nz,nx;
} sharedchunk_s1;

typedef struct {
  int nz,nx;
} numblocks_s1;
//!ACC$REGION(<50:chunk>,<nx/50>,<nz,nx:in>,<operand:none>,<phys2d:inout>,<operand:none,shared>) BEGIN
__global__ void sharedchunk_Kernel1(sharedchunk_s1 sharedchunk_k1,int *operand,int *phys2d) {
  int nz = sharedchunk_k1.nz;
  int nx = sharedchunk_k1.nx;

  int iter;
  int k;
  int i;
  for (iter=1;iter<=200;iter++) {
    for (k=1;k<=nz;k++) {
//!ACC$DO PARALLEL(1)
      i = (blockIdx.x*blockDim.x)+threadIdx.x+1;
//      for (i=1;i<=nx;i++) {
        operandS[FTNREF2D(i,k,50,blockIdx.x*blockDim.x+1,1)] = (i - 1) * nz + k;
//      }
    }
    for (k=1;k<=nz;k++) {
//!ACC$DO PARALLEL(1)
      i = (blockIdx.x*blockDim.x)+threadIdx.x+1;
//      for (i=1;i<=nx;i++) {
        operandS[FTNREF2D(i,k,50,blockIdx.x*blockDim.x+1,1)] = operandS[FTNREF2D(i,k,50,blockIdx.x*blockDim.x+1,1)] * 2;
//      }
    }
    for (k=1;k<=nz;k++) {
//!ACC$DO PARALLEL(1)
      i = (blockIdx.x*blockDim.x)+threadIdx.x+1;
//      for (i=1;i<=nx;i++) {
        operandS[FTNREF2D(i,k,50,blockIdx.x*blockDim.x+1,1)] = operandS[FTNREF2D(i,k,50,blockIdx.x*blockDim.x+1,1)] * 2;
//      }
    }
    for (k=1;k<=nz;k++) {
//!ACC$DO PARALLEL(1)
      i = (blockIdx.x*blockDim.x)+threadIdx.x+1;
//      for (i=1;i<=nx;i++) {
        operandS[FTNREF2D(i,k,50,blockIdx.x*blockDim.x+1,1)] = operandS[FTNREF2D(i,k,50,blockIdx.x*blockDim.x+1,1)] / 2;
//      }
    }
    for (k=1;k<=nz;k++) {
//!ACC$DO PARALLEL(1)
      i = (blockIdx.x*blockDim.x)+threadIdx.x+1;
//      for (i=1;i<=nx;i++) {
        operandS[FTNREF2D(i,k,50,blockIdx.x*blockDim.x+1,1)] = operandS[FTNREF2D(i,k,50,blockIdx.x*blockDim.x+1,1)] / 2;
//      }
    }
    for (k=1;k<=nz;k++) {
//!ACC$DO PARALLEL(1)
      i = (blockIdx.x*blockDim.x)+threadIdx.x+1;
//      for (i=1;i<=nx;i++) {
        phys2d[FTNREF2D(i,k,nx,1,1)] = operandS[FTNREF2D(i,k,50,blockIdx.x*blockDim.x+1,1)];
//      }
    }
  }

  return;
}
//!ACC$REGION END
//!ACC$REGION(<nx:block=2>,<nz/2>,<nz,nx:in>,<phys2d:out>) BEGIN
__global__ void numblocks_Kernel1(numblocks_s1 numblocks_k1,int *phys2d) {
  int nz = numblocks_k1.nz;
  int nx = numblocks_k1.nx;

  int k;
  int i;
//!ACC$DO PARALLEL (1)
  k = (blockIdx.x*2)+(threadIdx.x/(blockDim.x/2))+1;
//  for (k=1;k<=nz;k++) {
//!ACC$DO VECTOR (1)
    i = threadIdx.x%(blockDim.x/2)+1;
//    for (i=1;i<=nx;i++) {
      phys2d[FTNREF2D(i,k,nx,1,1)] = k * nx + nz;
//    }
//  }

  return;
}
//!ACC$REGION END
extern "C" void sharedchunk_ (int *nx__G,int *nz__G,int *phys2d) {

  static float runtime = 0.0;
  float elapsed_time_ms = 0.f;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  int nx=*nx__G;
  int nz=*nz__G;
hipError_t hipError_t;
  int *D_operand;
  hipError_t = hipMalloc((void **) &D_operand,((nx)*(nz))*sizeof(int));
  cudaErrorChk("sharedChunk.f90.ACC.1",11,"operand",hipError_t);
  int *D_phys2d;
  hipError_t = hipMalloc((void **) &D_phys2d,((nx)*(nz))*sizeof(int));
  cudaErrorChk("sharedChunk.f90.ACC.1",11,"phys2d",hipError_t);
//!ACC$DATA(<operand:none,shared(50,50)>)
//!MWG: F2C-ACC V4.7, 4.8 bug which requires <operand:none> appear before the
//!MWG: tuple does.

  dim3 cuda_threads1(50);
  dim3 cuda_grids1(nx/50);

  unsigned int timer = 0;

  sharedchunk_s1 sharedchunk_k1;
  sharedchunk_k1.nz = nz;
  sharedchunk_k1.nx = nx;
  hipError_t = hipMemcpy(D_phys2d,phys2d,((nx)*(nz))*sizeof(int),hipMemcpyHostToDevice);
  cudaErrorChk("sharedChunk.f90.ACC.1",11,"phys2d",hipError_t);

  hipEventRecord( start, 0 );
  hipEventSynchronize(start);
  // setup execution parameters 
  // execute the kernel 
  sharedchunk_Kernel1<<< cuda_grids1, cuda_threads1 >>>(sharedchunk_k1,D_operand,D_phys2d);
  hipDeviceSynchronize();

  hipEventRecord( stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime( &elapsed_time_ms, start, stop );
  runtime += elapsed_time_ms;
  //  printf("sharedchunk kernel execution time: %.2f ms\n",runtime);
  hipError_t = hipMemcpy(phys2d,D_phys2d,((nx)*(nz))*sizeof(int),hipMemcpyDeviceToHost);
  cudaErrorChk("sharedChunk.f90.ACC.1",50,"phys2d",hipError_t);
  hipError_t = hipFree(D_operand);
  cudaErrorChk("sharedChunk.f90.ACC.1",51,"operand",hipError_t);
  hipError_t = hipFree(D_phys2d);
  cudaErrorChk("sharedChunk.f90.ACC.1",51,"phys2d",hipError_t);

  return;
}
//! Routine to test the F2C-ACC ability to assign multiple columns to a thread block.
extern "C" void numblocks_ (int *nx__G,int *nz__G,int *phys2d) {

  static float runtime = 0.0;
  float elapsed_time_ms = 0.f;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  int nx=*nx__G;
  int nz=*nz__G;
hipError_t hipError_t;
  int *D_phys2d;
  hipError_t = hipMalloc((void **) &D_phys2d,((nx)*(nz))*sizeof(int));
  cudaErrorChk("sharedChunk.f90.ACC.1",59,"phys2d",hipError_t);


  dim3 cuda_threads1((nx)*2);
  dim3 cuda_grids1((nz/2)/2);

  unsigned int timer = 0;

  numblocks_s1 numblocks_k1;
  numblocks_k1.nz = nz;
  numblocks_k1.nx = nx;

  hipEventRecord( start, 0 );
  hipEventSynchronize(start);
  // setup execution parameters 
  // execute the kernel 
  numblocks_Kernel1<<< cuda_grids1, cuda_threads1 >>>(numblocks_k1,D_phys2d);
  hipDeviceSynchronize();

  hipEventRecord( stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime( &elapsed_time_ms, start, stop );
  runtime += elapsed_time_ms;
  //  printf("numblocks kernel execution time: %.2f ms\n",runtime);
  hipError_t = hipMemcpy(phys2d,D_phys2d,((nx)*(nz))*sizeof(int),hipMemcpyDeviceToHost);
  cudaErrorChk("sharedChunk.f90.ACC.1",67,"phys2d",hipError_t);
  hipError_t = hipFree(D_phys2d);
  cudaErrorChk("sharedChunk.f90.ACC.1",68,"phys2d",hipError_t);

  return;
}
