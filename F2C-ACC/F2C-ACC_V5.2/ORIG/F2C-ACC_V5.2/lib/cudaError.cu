
#include <hip/hip_runtime.h>
#include <stdio.h>

void cudaErrorChk(char *filename, int line, char *varname, hipError_t hipError_t) {
  if (hipError_t != hipSuccess) {
    printf("%s:%d variable: \"%s\" CUDA error: %s\n",filename,line,varname,hipGetErrorString(hipError_t));
    exit(-1);
  }
  return;
}


