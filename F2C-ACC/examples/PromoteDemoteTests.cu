#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "ftocmacros.h"
#include "F2Cinclude.h"
#define TRUE 1
#define FALSE 0
#define NZ 50
#define NX 5000
#define NZ 50
#define NX 5000
   __device__ int operandP[(NZ)*(NX)];
typedef struct {
  int nz,nx;
} global1dpromote2dblock_s1;

   __device__ int operandtP[(NX)*(NX)];
typedef struct {
  int nz,nx;
} global1dpromote2dthread_s1;
//!ACC$REGION(<nz>,<nx>,<nz,nx:in>,<bigarray:inout>,<operand:none,global,promote(1:block)>) BEGIN
__global__ void global1dpromote2dblock_Kernel1(global1dpromote2dblock_s1 global1dpromote2dblock_k1,int *bigarray,int *operand) {
  int nz = global1dpromote2dblock_k1.nz;
  int nx = global1dpromote2dblock_k1.nx;

  int iter;
  int i;
  int k;
  for (iter=1;iter<=500;iter++) {
//!ACC$DO PARALLEL(1)
    i = blockIdx.x+1;
//    for (i=1;i<=nx;i++) {
//!ACC$DO VECTOR(1)
      k = threadIdx.x+1;
//      for (k=1;k<=nz;k++) {
        operandP[FTNREF2D(k,blockIdx.x,nz,1,0)] = i;
        operandP[FTNREF2D(k,blockIdx.x,nz,1,0)] = operandP[FTNREF2D(k,blockIdx.x,nz,1,0)] * 2;
        operandP[FTNREF2D(k,blockIdx.x,nz,1,0)] = operandP[FTNREF2D(k,blockIdx.x,nz,1,0)] * 2;
        operandP[FTNREF2D(k,blockIdx.x,nz,1,0)] = operandP[FTNREF2D(k,blockIdx.x,nz,1,0)] / 2;
        bigarray[FTNREF2D(k,i,nz,1,1)] = operandP[FTNREF2D(k,blockIdx.x,nz,1,0)] / 2;
//      }
//    }
  }

  return;
}
//!ACC$REGION END
//!ACC$REGION(<nz:block=2>,<nx>,<nz,nx:in>,<bigarray:inout>,<operandt:none,global,promote(1:block)>) BEGIN
__global__ void global1dpromote2dthread_Kernel1(global1dpromote2dthread_s1 global1dpromote2dthread_k1,int *bigarray,int *operandt) {
  int nz = global1dpromote2dthread_k1.nz;
  int nx = global1dpromote2dthread_k1.nx;

  int iter;
  int i;
  int k;
  for (iter=1;iter<=500;iter++) {
//!ACC$DO PARALLEL(1)
    i = (blockIdx.x*2)+(threadIdx.x/(blockDim.x/2))+1;
//    for (i=1;i<=nx;i++) {
      operandtP[FTNREF2D(i,(blockIdx.x*2)+(threadIdx.x/(blockDim.x/2)),nx,1,0)] = i;
      operandtP[FTNREF2D(i,(blockIdx.x*2)+(threadIdx.x/(blockDim.x/2)),nx,1,0)] = operandtP[FTNREF2D(i,(blockIdx.x*2)+(threadIdx.x/(blockDim.x/2)),nx,1,0)] * 2;
      operandtP[FTNREF2D(i,(blockIdx.x*2)+(threadIdx.x/(blockDim.x/2)),nx,1,0)] = operandtP[FTNREF2D(i,(blockIdx.x*2)+(threadIdx.x/(blockDim.x/2)),nx,1,0)] * 2;
      operandtP[FTNREF2D(i,(blockIdx.x*2)+(threadIdx.x/(blockDim.x/2)),nx,1,0)] = operandtP[FTNREF2D(i,(blockIdx.x*2)+(threadIdx.x/(blockDim.x/2)),nx,1,0)] / 2;
//!ACC$DO VECTOR(1)
      k = threadIdx.x%(blockDim.x/2)+1;
//      for (k=1;k<=nz;k++) {
        bigarray[FTNREF2D(k,i,nz,1,1)] = operandtP[FTNREF2D(i,(blockIdx.x*2)+(threadIdx.x/(blockDim.x/2)),nx,1,0)] / 2;
//      }
//    }
  }

  return;
}
//!ACC$REGION END
//! Unit test for GPU execution
//! Written by:      	Mark Govett
//! Date:      		February 2012
//!
//!Unit tests to demonstrate the ability to promote and demote variables
//!which may be required for correctness (promote) or performance (shared)
//!   global1Dpromote2Dblock:      routine to promote the block dimension
//!   global1Dpromote2Dthread:      routine to promote the thread dimension
//!
extern "C" void global1dpromote2dblock_ (int *nx__G,int *nz__G,int *bigarray) {

  static float runtime = 0.0;
  float elapsed_time_ms = 0.f;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  int nx=*nx__G;
  int nz=*nz__G;
hipError_t hipError_t;
  int *D_bigarray;
  hipError_t = hipMalloc((void **) &D_bigarray,((nz)*(nx))*sizeof(int));
  cudaErrorChk("PromoteDemoteTests.f90.ACC.1",17,"bigarray",hipError_t);
  int *D_operand;
  hipError_t = hipMalloc((void **) &D_operand,((nz))*sizeof(int));
  cudaErrorChk("PromoteDemoteTests.f90.ACC.1",17,"operand",hipError_t);

  int *operand = (int*) malloc((nz)*sizeof(int));
  if (operand == NULL) {
    printf("ERROR: memory allocation error");
    exit(1);
  }


  dim3 cuda_threads1(NZ);
  dim3 cuda_grids1(NX);

  unsigned int timer = 0;

  global1dpromote2dblock_s1 global1dpromote2dblock_k1;
  global1dpromote2dblock_k1.nz = nz;
  global1dpromote2dblock_k1.nx = nx;
  hipError_t = hipMemcpy(D_bigarray,bigarray,((nz)*(nx))*sizeof(int),hipMemcpyHostToDevice);
  cudaErrorChk("PromoteDemoteTests.f90.ACC.1",17,"bigarray",hipError_t);

  hipEventRecord( start, 0 );
  hipEventSynchronize(start);
  // setup execution parameters 
  // execute the kernel 
  global1dpromote2dblock_Kernel1<<< cuda_grids1, cuda_threads1 >>>(global1dpromote2dblock_k1,D_bigarray,D_operand);
  hipDeviceSynchronize();

  hipEventRecord( stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime( &elapsed_time_ms, start, stop );
  runtime += elapsed_time_ms;
  //  printf("global1dpromote2dblock kernel execution time: %.2f ms\n",runtime);
  hipError_t = hipMemcpy(bigarray,D_bigarray,((nz)*(nx))*sizeof(int),hipMemcpyDeviceToHost);
  cudaErrorChk("PromoteDemoteTests.f90.ACC.1",31,"bigarray",hipError_t);
  hipError_t = hipFree(D_bigarray);
  cudaErrorChk("PromoteDemoteTests.f90.ACC.1",32,"bigarray",hipError_t);
  free(operand);
  hipError_t = hipFree(D_operand);
  cudaErrorChk("PromoteDemoteTests.f90.ACC.1",32,"operand",hipError_t);

  return;
}
//! routine to promote the thread dimension
extern "C" void global1dpromote2dthread_ (int *nx__G,int *nz__G,int *bigarray) {

  static float runtime = 0.0;
  float elapsed_time_ms = 0.f;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  int nx=*nx__G;
  int nz=*nz__G;
hipError_t hipError_t;
  int *D_bigarray;
  hipError_t = hipMalloc((void **) &D_bigarray,((nz)*(nx))*sizeof(int));
  cudaErrorChk("PromoteDemoteTests.f90.ACC.1",42,"bigarray",hipError_t);
  int *D_operandt;
  hipError_t = hipMalloc((void **) &D_operandt,((nx))*sizeof(int));
  cudaErrorChk("PromoteDemoteTests.f90.ACC.1",42,"operandt",hipError_t);

  int *operandt = (int*) malloc((nx)*sizeof(int));
  if (operandt == NULL) {
    printf("ERROR: memory allocation error");
    exit(1);
  }


  dim3 cuda_threads1((NZ)*2);
  dim3 cuda_grids1((NX)/2);

  unsigned int timer = 0;

  global1dpromote2dthread_s1 global1dpromote2dthread_k1;
  global1dpromote2dthread_k1.nz = nz;
  global1dpromote2dthread_k1.nx = nx;
  hipError_t = hipMemcpy(D_bigarray,bigarray,((nz)*(nx))*sizeof(int),hipMemcpyHostToDevice);
  cudaErrorChk("PromoteDemoteTests.f90.ACC.1",42,"bigarray",hipError_t);

  hipEventRecord( start, 0 );
  hipEventSynchronize(start);
  // setup execution parameters 
  // execute the kernel 
  global1dpromote2dthread_Kernel1<<< cuda_grids1, cuda_threads1 >>>(global1dpromote2dthread_k1,D_bigarray,D_operandt);
  hipDeviceSynchronize();

  hipEventRecord( stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime( &elapsed_time_ms, start, stop );
  runtime += elapsed_time_ms;
  //  printf("global1dpromote2dthread kernel execution time: %.2f ms\n",runtime);
  hipError_t = hipMemcpy(bigarray,D_bigarray,((nz)*(nx))*sizeof(int),hipMemcpyDeviceToHost);
  cudaErrorChk("PromoteDemoteTests.f90.ACC.1",56,"bigarray",hipError_t);
  hipError_t = hipFree(D_bigarray);
  cudaErrorChk("PromoteDemoteTests.f90.ACC.1",57,"bigarray",hipError_t);
  free(operandt);
  hipError_t = hipFree(D_operandt);
  cudaErrorChk("PromoteDemoteTests.f90.ACC.1",57,"operandt",hipError_t);

  return;
}
