
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "ftocmacros.h"
#include "F2Cinclude.h"
#define TRUE 1
#define FALSE 0
int *D_nz, *H_nz;
int *D_nip, *H_nip;
float *D_vol, *H_vol;
//! Unit tests for ACC$REGION Data handling
//! Designed for GPU translation and execution
//! Mark Govett       Nov 2010
//! NOAA/ESRL
//!
//! ACC$DATA is used to copy data between GPU and CPU, and permit other
//! routines to access that data (without requiring a copy)
//!
//! this is a no-op routine for the CPU
//!
extern "C" void copyin_ (int *nz__G,int *nip__G,float *vol,float *factor__G) {

  int nz=*nz__G;
  int nip=*nip__G;
  float factor=*factor__G;
hipError_t hipError_t;
//! "in" specifies data will be copied to the GPU
//! "global" specifies the pointer defined to access the data will have global scope
//!
//!ACC$DATA(<nz,nip,vol:in,global>)
  hipError_t = hipMalloc((void **) &D_nz,(1)*sizeof(int));
  cudaErrorChk("copyIn.f90.ACC.1",19,"nz",hipError_t);
  hipError_t = hipMemcpy(D_nz,&nz,(1)*sizeof(int),hipMemcpyHostToDevice);
  cudaErrorChk("copyIn.f90.ACC.1",19,"nz",hipError_t);
  H_nz = nz__G;
  hipError_t = hipMalloc((void **) &D_nip,(1)*sizeof(int));
  cudaErrorChk("copyIn.f90.ACC.1",19,"nip",hipError_t);
  hipError_t = hipMemcpy(D_nip,&nip,(1)*sizeof(int),hipMemcpyHostToDevice);
  cudaErrorChk("copyIn.f90.ACC.1",19,"nip",hipError_t);
  H_nip = nip__G;
  hipError_t = hipMalloc((void **) &D_vol,((nz)*(nip))*sizeof(float));
  cudaErrorChk("copyIn.f90.ACC.1",19,"vol",hipError_t);
  hipError_t = hipMemcpy(D_vol,vol,((nz)*(nip))*sizeof(float),hipMemcpyHostToDevice);
  cudaErrorChk("copyIn.f90.ACC.1",19,"vol",hipError_t);
  H_vol = vol;

}
