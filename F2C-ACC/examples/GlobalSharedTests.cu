#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "ftocmacros.h"
#include "F2Cinclude.h"
#define TRUE 1
#define FALSE 0
typedef struct {
  int nz,nx;
} global2d_s1;

   __shared__ int operandS[50];
typedef struct {
  int nz,nx;
} global2dshared1d_s1;

typedef struct {
  int nz,nx;
} global2dloops_s1;

typedef struct {
  int nz,nx;
} global2dshared1dloops_s1;

typedef struct {
  int nz,nx,ny;
} global3d_s1;

   __shared__ int operand3dS[50];
typedef struct {
  int nz,nx,ny;
} global3dshared1d_s1;
//!ACC$REGION(<nz>,<nx>,<operand:none>,<nz,nx:in>,<dyn2d:inout>) BEGIN
__global__ void global2d_Kernel1(int *operand,global2d_s1 global2d_k1,int *dyn2d) {
  int nz = global2d_k1.nz;
  int nx = global2d_k1.nx;

  int iter;
  int i;
  int k;
  for (iter=1;iter<=200;iter++) {
//!ACC$DO PARALLEL(1)
    i = blockIdx.x+1;
//    for (i=1;i<=nx;i++) {
//!ACC$DO VECTOR(1)
      k = threadIdx.x+1;
//      for (k=1;k<=nz;k++) {
        operand[FTNREF2D(k,i,nz,1,1)] = (i - 1) * nx + k;
        operand[FTNREF2D(k,i,nz,1,1)] = operand[FTNREF2D(k,i,nz,1,1)] * 2;
        operand[FTNREF2D(k,i,nz,1,1)] = operand[FTNREF2D(k,i,nz,1,1)] * 2;
        operand[FTNREF2D(k,i,nz,1,1)] = operand[FTNREF2D(k,i,nz,1,1)] / 2;
        operand[FTNREF2D(k,i,nz,1,1)] = operand[FTNREF2D(k,i,nz,1,1)] / 2;
        dyn2d[FTNREF2D(k,i,nz,1,1)] = operand[FTNREF2D(k,i,nz,1,1)];
//      }
//    }
  }

  return;
}
//!ACC$REGION END
//!ACC$REGION(<nz>,<nx>,<operand:none>,<nz,nx:in>,<dyn2d:inout>,<operand:none,shared>) BEGIN
__global__ void global2dshared1d_Kernel1(int *operand,global2dshared1d_s1 global2dshared1d_k1,int *dyn2d) {
  int nz = global2dshared1d_k1.nz;
  int nx = global2dshared1d_k1.nx;

  int iter;
  int i;
  int k;
  for (iter=1;iter<=200;iter++) {
//!ACC$DO PARALLEL(1)
    i = blockIdx.x+1;
//    for (i=1;i<=nx;i++) {
//!ACC$DO VECTOR(1)
      k = threadIdx.x+1;
//      for (k=1;k<=nz;k++) {
        operandS[FTNREF1D(k,1)] = (i - 1) * nx + k;
        operandS[FTNREF1D(k,1)] = operandS[FTNREF1D(k,1)] * 2;
        operandS[FTNREF1D(k,1)] = operandS[FTNREF1D(k,1)] * 2;
        operandS[FTNREF1D(k,1)] = operandS[FTNREF1D(k,1)] / 2;
        operandS[FTNREF1D(k,1)] = operandS[FTNREF1D(k,1)] / 2;
        dyn2d[FTNREF2D(k,i,nz,1,1)] = operandS[FTNREF1D(k,1)];
//      }
//    }
  }

  return;
}
//!ACC$REGION END
//!ACC$REGION(<nz>,<nx>,<operand:none>,<nz,nx:in>,<dyn2d:inout>) BEGIN
__global__ void global2dloops_Kernel1(int *operand,global2dloops_s1 global2dloops_k1,int *dyn2d) {
  int nz = global2dloops_k1.nz;
  int nx = global2dloops_k1.nx;

  int iter;
  int i;
  int k;
  for (iter=1;iter<=200;iter++) {
//!ACC$DO PARALLEL(1)
    i = blockIdx.x+1;
//    for (i=1;i<=nx;i++) {
//!ACC$DO VECTOR(1)
      k = threadIdx.x+1;
//      for (k=1;k<=nz;k++) {
        operand[FTNREF2D(k,i,nz,1,1)] = (i - 1) * nx + k;
//      }
//    }
//!ACC$DO PARALLEL(1)
    i = blockIdx.x+1;
//    for (i=1;i<=nx;i++) {
//!ACC$DO VECTOR(1)
      k = threadIdx.x+1;
//      for (k=1;k<=nz;k++) {
        operand[FTNREF2D(k,i,nz,1,1)] = operand[FTNREF2D(k,i,nz,1,1)] * 2;
//      }
//    }
//!ACC$DO PARALLEL(1)
    i = blockIdx.x+1;
//    for (i=1;i<=nx;i++) {
//!ACC$DO VECTOR(1)
      k = threadIdx.x+1;
//      for (k=1;k<=nz;k++) {
        operand[FTNREF2D(k,i,nz,1,1)] = operand[FTNREF2D(k,i,nz,1,1)] * 2;
//      }
//    }
//!ACC$DO PARALLEL(1)
    i = blockIdx.x+1;
//    for (i=1;i<=nx;i++) {
//!ACC$DO VECTOR(1)
      k = threadIdx.x+1;
//      for (k=1;k<=nz;k++) {
        operand[FTNREF2D(k,i,nz,1,1)] = operand[FTNREF2D(k,i,nz,1,1)] / 2;
//      }
//    }
//!ACC$DO PARALLEL(1)
    i = blockIdx.x+1;
//    for (i=1;i<=nx;i++) {
//!ACC$DO VECTOR(1)
      k = threadIdx.x+1;
//      for (k=1;k<=nz;k++) {
        operand[FTNREF2D(k,i,nz,1,1)] = operand[FTNREF2D(k,i,nz,1,1)] / 2;
//      }
//    }
//!ACC$DO PARALLEL(1)
    i = blockIdx.x+1;
//    for (i=1;i<=nx;i++) {
//!ACC$DO VECTOR(1)
      k = threadIdx.x+1;
//      for (k=1;k<=nz;k++) {
        dyn2d[FTNREF2D(k,i,nz,1,1)] = operand[FTNREF2D(k,i,nz,1,1)];
//      }
//    }
  }

  return;
}
//!ACC$REGION END
//!ACC$REGION(<nz>,<nx>,<operand:none>,<nz,nx:in>,<dyn2d:inout>,<operand:none,shared,demote(2)>) BEGIN
__global__ void global2dshared1dloops_Kernel1(int *operand,global2dshared1dloops_s1 global2dshared1dloops_k1,int *dyn2d) {
  int nz = global2dshared1dloops_k1.nz;
  int nx = global2dshared1dloops_k1.nx;

  int iter;
  int i;
  int k;
  for (iter=1;iter<=200;iter++) {
//!ACC$DO PARALLEL(1)
    i = blockIdx.x+1;
//    for (i=1;i<=nx;i++) {
//!ACC$DO VECTOR(1)
      k = threadIdx.x+1;
//      for (k=1;k<=nz;k++) {
        operandS[FTNREF1D(k,1)] = (i - 1) * nx + k;
//      }
//    }
//!ACC$DO PARALLEL(1)
    i = blockIdx.x+1;
//    for (i=1;i<=nx;i++) {
//!ACC$DO VECTOR(1)
      k = threadIdx.x+1;
//      for (k=1;k<=nz;k++) {
        operandS[FTNREF1D(k,1)] = operandS[FTNREF1D(k,1)] * 2;
//      }
//    }
//!ACC$DO PARALLEL(1)
    i = blockIdx.x+1;
//    for (i=1;i<=nx;i++) {
//!ACC$DO VECTOR(1)
      k = threadIdx.x+1;
//      for (k=1;k<=nz;k++) {
        operandS[FTNREF1D(k,1)] = operandS[FTNREF1D(k,1)] * 2;
//      }
//    }
//!ACC$DO PARALLEL(1)
    i = blockIdx.x+1;
//    for (i=1;i<=nx;i++) {
//!ACC$DO VECTOR(1)
      k = threadIdx.x+1;
//      for (k=1;k<=nz;k++) {
        operandS[FTNREF1D(k,1)] = operandS[FTNREF1D(k,1)] / 2;
//      }
//    }
//!ACC$DO PARALLEL(1)
    i = blockIdx.x+1;
//    for (i=1;i<=nx;i++) {
//!ACC$DO VECTOR(1)
      k = threadIdx.x+1;
//      for (k=1;k<=nz;k++) {
        operandS[FTNREF1D(k,1)] = operandS[FTNREF1D(k,1)] / 2;
//      }
//    }
//!ACC$DO PARALLEL(1)
    i = blockIdx.x+1;
//    for (i=1;i<=nx;i++) {
//!ACC$DO VECTOR(1)
      k = threadIdx.x+1;
//      for (k=1;k<=nz;k++) {
        dyn2d[FTNREF2D(k,i,nz,1,1)] = operandS[FTNREF1D(k,1)];
//      }
//    }
  }

  return;
}
//!ACC$REGION END
//!ACC$REGION(<nz>,<nx,ny>,<operand:none>,<nz,nx,ny:in>,<dyn3d:out>) BEGIN
__global__ void global3d_Kernel1(int *operand,global3d_s1 global3d_k1,int *dyn3d) {
  int nz = global3d_k1.nz;
  int nx = global3d_k1.nx;
  int ny = global3d_k1.ny;

  int iter;
  int i;
  int j;
  int k;
  for (iter=1;iter<=200;iter++) {
//!ACC$DO PARALLEL(1)
    i = blockIdx.x+1;
//    for (i=1;i<=nx;i++) {
//!ACC$DO PARALLEL(2)
      j = blockIdx.y+1;
//      for (j=1;j<=ny;j++) {
//!ACC$DO VECTOR(1)
        k = threadIdx.x+1;
//        for (k=1;k<=nz;k++) {
          operand[FTNREF3D(k,i,j,nz,nx,1,1,1)] = ((i - 1) * nx + j) * k;
//        }
//      }
//    }
//!ACC$DO PARALLEL(1)
    i = blockIdx.x+1;
//    for (i=1;i<=nx;i++) {
//!ACC$DO PARALLEL(2)
      j = blockIdx.y+1;
//      for (j=1;j<=ny;j++) {
//!ACC$DO VECTOR(1)
        k = threadIdx.x+1;
//        for (k=1;k<=nz;k++) {
          operand[FTNREF3D(k,i,j,nz,nx,1,1,1)] = operand[FTNREF3D(k,i,j,nz,nx,1,1,1)] * 2;
//        }
//      }
//    }
//!ACC$DO PARALLEL(1)
    i = blockIdx.x+1;
//    for (i=1;i<=nx;i++) {
//!ACC$DO PARALLEL(2)
      j = blockIdx.y+1;
//      for (j=1;j<=ny;j++) {
//!ACC$DO VECTOR(1)
        k = threadIdx.x+1;
//        for (k=1;k<=nz;k++) {
          operand[FTNREF3D(k,i,j,nz,nx,1,1,1)] = operand[FTNREF3D(k,i,j,nz,nx,1,1,1)] * 2;
//        }
//      }
//    }
//!ACC$DO PARALLEL(1)
    i = blockIdx.x+1;
//    for (i=1;i<=nx;i++) {
//!ACC$DO PARALLEL(2)
      j = blockIdx.y+1;
//      for (j=1;j<=ny;j++) {
//!ACC$DO VECTOR(1)
        k = threadIdx.x+1;
//        for (k=1;k<=nz;k++) {
          operand[FTNREF3D(k,i,j,nz,nx,1,1,1)] = operand[FTNREF3D(k,i,j,nz,nx,1,1,1)] / 2;
//        }
//      }
//    }
//!ACC$DO PARALLEL(1)
    i = blockIdx.x+1;
//    for (i=1;i<=nx;i++) {
//!ACC$DO PARALLEL(2)
      j = blockIdx.y+1;
//      for (j=1;j<=ny;j++) {
//!ACC$DO VECTOR(1)
        k = threadIdx.x+1;
//        for (k=1;k<=nz;k++) {
          operand[FTNREF3D(k,i,j,nz,nx,1,1,1)] = operand[FTNREF3D(k,i,j,nz,nx,1,1,1)] / 2;
//        }
//      }
//    }
//!ACC$DO PARALLEL(1)
    i = blockIdx.x+1;
//    for (i=1;i<=nx;i++) {
//!ACC$DO PARALLEL(2)
      j = blockIdx.y+1;
//      for (j=1;j<=ny;j++) {
//!ACC$DO VECTOR(1)
        k = threadIdx.x+1;
//        for (k=1;k<=nz;k++) {
          dyn3d[FTNREF3D(k,i,j,nz,nx,1,1,1)] = operand[FTNREF3D(k,i,j,nz,nx,1,1,1)];
//        }
//      }
//    }
  }

  return;
}
//!ACC$REGION END
//!ACC$REGION(<nz>,<nx,ny>,<operand3d:none>,<nz,nx,ny:in>,<dyn3d:out>,<operand3d:none,shared,demote(2,3)>) BEGIN
__global__ void global3dshared1d_Kernel1(int *operand3d,global3dshared1d_s1 global3dshared1d_k1,int *dyn3d) {
  int nz = global3dshared1d_k1.nz;
  int nx = global3dshared1d_k1.nx;
  int ny = global3dshared1d_k1.ny;

  int iter;
  int i;
  int j;
  int k;
  for (iter=1;iter<=200;iter++) {
//!ACC$DO PARALLEL(1)
    i = blockIdx.x+1;
//    for (i=1;i<=nx;i++) {
//!ACC$DO PARALLEL(2)
      j = blockIdx.y+1;
//      for (j=1;j<=ny;j++) {
//!ACC$DO VECTOR(1)
        k = threadIdx.x+1;
//        for (k=1;k<=nz;k++) {
          operand3dS[FTNREF1D(k,1)] = ((i - 1) * nx + j) * k;
//        }
//      }
//    }
//!ACC$DO PARALLEL(1)
    i = blockIdx.x+1;
//    for (i=1;i<=nx;i++) {
//!ACC$DO PARALLEL(2)
      j = blockIdx.y+1;
//      for (j=1;j<=ny;j++) {
//!ACC$DO VECTOR(1)
        k = threadIdx.x+1;
//        for (k=1;k<=nz;k++) {
          operand3dS[FTNREF1D(k,1)] = operand3dS[FTNREF1D(k,1)] * 2;
//        }
//      }
//    }
//!ACC$DO PARALLEL(1)
    i = blockIdx.x+1;
//    for (i=1;i<=nx;i++) {
//!ACC$DO PARALLEL(2)
      j = blockIdx.y+1;
//      for (j=1;j<=ny;j++) {
//!ACC$DO VECTOR(1)
        k = threadIdx.x+1;
//        for (k=1;k<=nz;k++) {
          operand3dS[FTNREF1D(k,1)] = operand3dS[FTNREF1D(k,1)] * 2;
//        }
//      }
//    }
//!ACC$DO PARALLEL(1)
    i = blockIdx.x+1;
//    for (i=1;i<=nx;i++) {
//!ACC$DO PARALLEL(2)
      j = blockIdx.y+1;
//      for (j=1;j<=ny;j++) {
//!ACC$DO VECTOR(1)
        k = threadIdx.x+1;
//        for (k=1;k<=nz;k++) {
          operand3dS[FTNREF1D(k,1)] = operand3dS[FTNREF1D(k,1)] / 2;
//        }
//      }
//    }
//!ACC$DO PARALLEL(1)
    i = blockIdx.x+1;
//    for (i=1;i<=nx;i++) {
//!ACC$DO PARALLEL(2)
      j = blockIdx.y+1;
//      for (j=1;j<=ny;j++) {
//!ACC$DO VECTOR(1)
        k = threadIdx.x+1;
//        for (k=1;k<=nz;k++) {
          operand3dS[FTNREF1D(k,1)] = operand3dS[FTNREF1D(k,1)] / 2;
//        }
//      }
//    }
//!ACC$DO PARALLEL(1)
    i = blockIdx.x+1;
//    for (i=1;i<=nx;i++) {
//!ACC$DO PARALLEL(2)
      j = blockIdx.y+1;
//      for (j=1;j<=ny;j++) {
//!ACC$DO VECTOR(1)
        k = threadIdx.x+1;
//        for (k=1;k<=nz;k++) {
          dyn3d[FTNREF3D(k,i,j,nz,nx,1,1,1)] = operand3dS[FTNREF1D(k,1)];
//        }
//      }
//    }
  }

  return;
}
//!ACC$REGION END
//! Performance and unit tests for the F2C-ACC compiler
//! Written by:      Mark Govett
//! Date:      	February 2012
//!
//! These tests compare performance using GPU global memory and shared memory
//! for a variety of cases as follows:
//!   global2D:      	baseline test using full 2D arrays for storage
//!   global2Dshared1D:      same as global2D but uses shared memory for "operand"
//!   global2Dloops:      same as baseline but each update appears in its own loop
//!   global2Dshared1Dloops:      shared memory test
//!global2D demonstrates repeated use of a single variable used to update another
//! variable within a single loop
extern "C" void global2d_ (int *nx__G,int *nz__G,int *dyn2d) {

  static float runtime = 0.0;
  float elapsed_time_ms = 0.f;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  int nx=*nx__G;
  int nz=*nz__G;
hipError_t hipError_t;
  int *D_operand;
  hipError_t = hipMalloc((void **) &D_operand,((nz)*(nx))*sizeof(int));
  cudaErrorChk("GlobalSharedTests.f90.ACC.1",20,"operand",hipError_t);
  int *D_dyn2d;
  hipError_t = hipMalloc((void **) &D_dyn2d,((nz)*(nx))*sizeof(int));
  cudaErrorChk("GlobalSharedTests.f90.ACC.1",20,"dyn2d",hipError_t);

  int *operand = (int*) malloc((nz)*(nx)*sizeof(int));
  if (operand == NULL) {
    printf("ERROR: memory allocation error");
    exit(1);
  }


  dim3 cuda_threads1(nz);
  dim3 cuda_grids1(nx);

  unsigned int timer = 0;

  global2d_s1 global2d_k1;
  global2d_k1.nz = nz;
  global2d_k1.nx = nx;
  hipError_t = hipMemcpy(D_dyn2d,dyn2d,((nz)*(nx))*sizeof(int),hipMemcpyHostToDevice);
  cudaErrorChk("GlobalSharedTests.f90.ACC.1",20,"dyn2d",hipError_t);
  hipError_t = hipMemcpy(D_dyn2d,dyn2d,((nz)*(nx))*sizeof(int),hipMemcpyHostToDevice);
  cudaErrorChk("GlobalSharedTests.f90.ACC.1",20,"dyn2d",hipError_t);

  hipEventRecord( start, 0 );
  hipEventSynchronize(start);
  // setup execution parameters 
  // execute the kernel 
  global2d_Kernel1<<< cuda_grids1, cuda_threads1 >>>(D_operand,global2d_k1,D_dyn2d);
  hipDeviceSynchronize();

  hipEventRecord( stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime( &elapsed_time_ms, start, stop );
  runtime += elapsed_time_ms;
  //  printf("global2d kernel execution time: %.2f ms\n",runtime);
  hipError_t = hipMemcpy(dyn2d,D_dyn2d,((nz)*(nx))*sizeof(int),hipMemcpyDeviceToHost);
  cudaErrorChk("GlobalSharedTests.f90.ACC.1",35,"dyn2d",hipError_t);
  free(operand);
  hipError_t = hipFree(D_operand);
  cudaErrorChk("GlobalSharedTests.f90.ACC.1",36,"operand",hipError_t);
  hipError_t = hipFree(D_dyn2d);
  cudaErrorChk("GlobalSharedTests.f90.ACC.1",36,"dyn2d",hipError_t);

  return;
}
extern "C" void global2dshared1d_ (int *nx__G,int *nz__G,int *dyn2d) {

  static float runtime = 0.0;
  float elapsed_time_ms = 0.f;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  int nx=*nx__G;
  int nz=*nz__G;
hipError_t hipError_t;
  int *D_operand;
  hipError_t = hipMalloc((void **) &D_operand,((nz))*sizeof(int));
  cudaErrorChk("GlobalSharedTests.f90.ACC.1",46,"operand",hipError_t);
  int *D_dyn2d;
  hipError_t = hipMalloc((void **) &D_dyn2d,((nz)*(nx))*sizeof(int));
  cudaErrorChk("GlobalSharedTests.f90.ACC.1",46,"dyn2d",hipError_t);
//!ACC$DATA(<operand:none,shared(50)>)


  dim3 cuda_threads1(nz);
  dim3 cuda_grids1(nx);

  unsigned int timer = 0;

  global2dshared1d_s1 global2dshared1d_k1;
  global2dshared1d_k1.nz = nz;
  global2dshared1d_k1.nx = nx;
  hipError_t = hipMemcpy(D_dyn2d,dyn2d,((nz)*(nx))*sizeof(int),hipMemcpyHostToDevice);
  cudaErrorChk("GlobalSharedTests.f90.ACC.1",46,"dyn2d",hipError_t);

  hipEventRecord( start, 0 );
  hipEventSynchronize(start);
  // setup execution parameters 
  // execute the kernel 
  global2dshared1d_Kernel1<<< cuda_grids1, cuda_threads1 >>>(D_operand,global2dshared1d_k1,D_dyn2d);
  hipDeviceSynchronize();

  hipEventRecord( stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime( &elapsed_time_ms, start, stop );
  runtime += elapsed_time_ms;
  //  printf("global2dshared1d kernel execution time: %.2f ms\n",runtime);
  hipError_t = hipMemcpy(dyn2d,D_dyn2d,((nz)*(nx))*sizeof(int),hipMemcpyDeviceToHost);
  cudaErrorChk("GlobalSharedTests.f90.ACC.1",61,"dyn2d",hipError_t);
  hipError_t = hipFree(D_operand);
  cudaErrorChk("GlobalSharedTests.f90.ACC.1",62,"operand",hipError_t);
  hipError_t = hipFree(D_dyn2d);
  cudaErrorChk("GlobalSharedTests.f90.ACC.1",62,"dyn2d",hipError_t);

  return;
}
extern "C" void global2dloops_ (int *nx__G,int *nz__G,int *dyn2d) {

  static float runtime = 0.0;
  float elapsed_time_ms = 0.f;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  int nx=*nx__G;
  int nz=*nz__G;
hipError_t hipError_t;
  int *D_operand;
  hipError_t = hipMalloc((void **) &D_operand,((nz)*(nx))*sizeof(int));
  cudaErrorChk("GlobalSharedTests.f90.ACC.1",71,"operand",hipError_t);
  int *D_dyn2d;
  hipError_t = hipMalloc((void **) &D_dyn2d,((nz)*(nx))*sizeof(int));
  cudaErrorChk("GlobalSharedTests.f90.ACC.1",71,"dyn2d",hipError_t);

  int *operand = (int*) malloc((nz)*(nx)*sizeof(int));
  if (operand == NULL) {
    printf("ERROR: memory allocation error");
    exit(1);
  }


  dim3 cuda_threads1(nz);
  dim3 cuda_grids1(nx);

  unsigned int timer = 0;

  global2dloops_s1 global2dloops_k1;
  global2dloops_k1.nz = nz;
  global2dloops_k1.nx = nx;
  hipError_t = hipMemcpy(D_dyn2d,dyn2d,((nz)*(nx))*sizeof(int),hipMemcpyHostToDevice);
  cudaErrorChk("GlobalSharedTests.f90.ACC.1",71,"dyn2d",hipError_t);
  hipError_t = hipMemcpy(D_dyn2d,dyn2d,((nz)*(nx))*sizeof(int),hipMemcpyHostToDevice);
  cudaErrorChk("GlobalSharedTests.f90.ACC.1",71,"dyn2d",hipError_t);

  hipEventRecord( start, 0 );
  hipEventSynchronize(start);
  // setup execution parameters 
  // execute the kernel 
  global2dloops_Kernel1<<< cuda_grids1, cuda_threads1 >>>(D_operand,global2dloops_k1,D_dyn2d);
  hipDeviceSynchronize();

  hipEventRecord( stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime( &elapsed_time_ms, start, stop );
  runtime += elapsed_time_ms;
  //  printf("global2dloops kernel execution time: %.2f ms\n",runtime);
  hipError_t = hipMemcpy(dyn2d,D_dyn2d,((nz)*(nx))*sizeof(int),hipMemcpyDeviceToHost);
  cudaErrorChk("GlobalSharedTests.f90.ACC.1",116,"dyn2d",hipError_t);
  free(operand);
  hipError_t = hipFree(D_operand);
  cudaErrorChk("GlobalSharedTests.f90.ACC.1",117,"operand",hipError_t);
  hipError_t = hipFree(D_dyn2d);
  cudaErrorChk("GlobalSharedTests.f90.ACC.1",117,"dyn2d",hipError_t);

  return;
}
extern "C" void global2dshared1dloops_ (int *nx__G,int *nz__G,int *dyn2d) {

  static float runtime = 0.0;
  float elapsed_time_ms = 0.f;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  int nx=*nx__G;
  int nz=*nz__G;
hipError_t hipError_t;
  int *D_operand;
  hipError_t = hipMalloc((void **) &D_operand,((nz)*(nx))*sizeof(int));
  cudaErrorChk("GlobalSharedTests.f90.ACC.1",130,"operand",hipError_t);
  int *D_dyn2d;
  hipError_t = hipMalloc((void **) &D_dyn2d,((nz)*(nx))*sizeof(int));
  cudaErrorChk("GlobalSharedTests.f90.ACC.1",130,"dyn2d",hipError_t);

  int *operand = (int*) malloc((nz)*(nx)*sizeof(int));
  if (operand == NULL) {
    printf("ERROR: memory allocation error");
    exit(1);
  }
//! MWG: this declaration is not needed because shared memory variables are
//! MWG: filebased declarations.  Since an ACC$DATA already declared this 
//! MWG: variable in an earlier routine, it is omitted here.
//!!ACC$DATA(<operand:none,shared(50)>)

  dim3 cuda_threads1(nz);
  dim3 cuda_grids1(nx);

  unsigned int timer = 0;

  global2dshared1dloops_s1 global2dshared1dloops_k1;
  global2dshared1dloops_k1.nz = nz;
  global2dshared1dloops_k1.nx = nx;
  hipError_t = hipMemcpy(D_dyn2d,dyn2d,((nz)*(nx))*sizeof(int),hipMemcpyHostToDevice);
  cudaErrorChk("GlobalSharedTests.f90.ACC.1",130,"dyn2d",hipError_t);

  hipEventRecord( start, 0 );
  hipEventSynchronize(start);
  // setup execution parameters 
  // execute the kernel 
  global2dshared1dloops_Kernel1<<< cuda_grids1, cuda_threads1 >>>(D_operand,global2dshared1dloops_k1,D_dyn2d);
  hipDeviceSynchronize();

  hipEventRecord( stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime( &elapsed_time_ms, start, stop );
  runtime += elapsed_time_ms;
  //  printf("global2dshared1dloops kernel execution time: %.2f ms\n",runtime);
  hipError_t = hipMemcpy(dyn2d,D_dyn2d,((nz)*(nx))*sizeof(int),hipMemcpyDeviceToHost);
  cudaErrorChk("GlobalSharedTests.f90.ACC.1",175,"dyn2d",hipError_t);
  free(operand);
  hipError_t = hipFree(D_operand);
  cudaErrorChk("GlobalSharedTests.f90.ACC.1",176,"operand",hipError_t);
  hipError_t = hipFree(D_dyn2d);
  cudaErrorChk("GlobalSharedTests.f90.ACC.1",176,"dyn2d",hipError_t);

  return;
}
extern "C" void global3d_ (int *nz__G,int *nx__G,int *ny__G,int *dyn3d) {

  static float runtime = 0.0;
  float elapsed_time_ms = 0.f;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  int nz=*nz__G;
  int nx=*nx__G;
  int ny=*ny__G;
hipError_t hipError_t;
  int *D_operand;
  hipError_t = hipMalloc((void **) &D_operand,((nz)*(nx)*(ny))*sizeof(int));
  cudaErrorChk("GlobalSharedTests.f90.ACC.1",185,"operand",hipError_t);
  int *D_dyn3d;
  hipError_t = hipMalloc((void **) &D_dyn3d,((nz)*(nx)*(ny))*sizeof(int));
  cudaErrorChk("GlobalSharedTests.f90.ACC.1",185,"dyn3d",hipError_t);

  int *operand = (int*) malloc((nz)*(nx)*(ny)*sizeof(int));
  if (operand == NULL) {
    printf("ERROR: memory allocation error");
    exit(1);
  }


  dim3 cuda_threads1(nz);
  dim3 cuda_grids1(nx,ny);

  unsigned int timer = 0;

  global3d_s1 global3d_k1;
  global3d_k1.nz = nz;
  global3d_k1.nx = nx;
  global3d_k1.ny = ny;

  hipEventRecord( start, 0 );
  hipEventSynchronize(start);
  // setup execution parameters 
  // execute the kernel 
  global3d_Kernel1<<< cuda_grids1, cuda_threads1 >>>(D_operand,global3d_k1,D_dyn3d);
  hipDeviceSynchronize();

  hipEventRecord( stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime( &elapsed_time_ms, start, stop );
  runtime += elapsed_time_ms;
  //  printf("global3d kernel execution time: %.2f ms\n",runtime);
  hipError_t = hipMemcpy(dyn3d,D_dyn3d,((nz)*(nx)*(ny))*sizeof(int),hipMemcpyDeviceToHost);
  cudaErrorChk("GlobalSharedTests.f90.ACC.1",248,"dyn3d",hipError_t);
  free(operand);
  hipError_t = hipFree(D_operand);
  cudaErrorChk("GlobalSharedTests.f90.ACC.1",249,"operand",hipError_t);
  hipError_t = hipFree(D_dyn3d);
  cudaErrorChk("GlobalSharedTests.f90.ACC.1",249,"dyn3d",hipError_t);

  return;
}
extern "C" void global3dshared1d_ (int *nz__G,int *nx__G,int *ny__G,int *dyn3d) {

  static float runtime = 0.0;
  float elapsed_time_ms = 0.f;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  int nz=*nz__G;
  int nx=*nx__G;
  int ny=*ny__G;
hipError_t hipError_t;
  int *D_operand3d;
  hipError_t = hipMalloc((void **) &D_operand3d,((nz)*(nx)*(ny))*sizeof(int));
  cudaErrorChk("GlobalSharedTests.f90.ACC.1",259,"operand3d",hipError_t);
  int *D_dyn3d;
  hipError_t = hipMalloc((void **) &D_dyn3d,((nz)*(nx)*(ny))*sizeof(int));
  cudaErrorChk("GlobalSharedTests.f90.ACC.1",259,"dyn3d",hipError_t);
//!ACC$DATA(<operand3D:none,shared(50)>)


  dim3 cuda_threads1(nz);
  dim3 cuda_grids1(nx,ny);

  unsigned int timer = 0;

  global3dshared1d_s1 global3dshared1d_k1;
  global3dshared1d_k1.nz = nz;
  global3dshared1d_k1.nx = nx;
  global3dshared1d_k1.ny = ny;

  hipEventRecord( start, 0 );
  hipEventSynchronize(start);
  // setup execution parameters 
  // execute the kernel 
  global3dshared1d_Kernel1<<< cuda_grids1, cuda_threads1 >>>(D_operand3d,global3dshared1d_k1,D_dyn3d);
  hipDeviceSynchronize();

  hipEventRecord( stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime( &elapsed_time_ms, start, stop );
  runtime += elapsed_time_ms;
  //  printf("global3dshared1d kernel execution time: %.2f ms\n",runtime);
  hipError_t = hipMemcpy(dyn3d,D_dyn3d,((nz)*(nx)*(ny))*sizeof(int),hipMemcpyDeviceToHost);
  cudaErrorChk("GlobalSharedTests.f90.ACC.1",322,"dyn3d",hipError_t);
  hipError_t = hipFree(D_operand3d);
  cudaErrorChk("GlobalSharedTests.f90.ACC.1",323,"operand3d",hipError_t);
  hipError_t = hipFree(D_dyn3d);
  cudaErrorChk("GlobalSharedTests.f90.ACC.1",323,"dyn3d",hipError_t);

  return;
}
