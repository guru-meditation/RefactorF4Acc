#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "ftocmacros.h"
#include "F2Cinclude.h"
#define TRUE 1
#define FALSE 0
//!ACC$REGION(<1>,<1>,<val,rval:out>) BEGIN
__global__ void intrinsics_Kernel1(int *D_val,float *D_rval) {
  int val = *D_val;
  float rval = *D_rval;

  int i;
  int j;
  int k;
  float fval;
  i = 3;
  j = 4;
  k = 5;
//! returns an integer
  val = MAX(fmaxf(i,j),k);
//! returns a float from integer arguments
  rval = MAX(fmaxf(i,j),k);
//!returns a float from float arguments
  fval = MAX(rval,1e8F);
  *D_val = val;
  *D_rval = rval;

  return;
}
//!ACC$REGION END
//!unit test for Fortran intrinsic functions (procedures)
//!Written by:      Mark Govett
//!Date:      	September 2011
//!
//! Designed for CPU and GPU runtime tests
//! CPU to test correctness
//! GPU to test function accuracy
//!
extern "C" void intrinsics_ (int *val__G,float *rval__G) {

  static float runtime = 0.0;
  float elapsed_time_ms = 0.f;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  int val=*val__G;
  float rval=*rval__G;
  int i;
  float fval;
hipError_t hipError_t;
  int *D_val;
  hipError_t = hipMalloc((void **) &D_val,(1)*sizeof(int));
  cudaErrorChk("intrinsics.f90.ACC.1",18,"val",hipError_t);
  float *D_rval;
  hipError_t = hipMalloc((void **) &D_rval,(1)*sizeof(float));
  cudaErrorChk("intrinsics.f90.ACC.1",18,"rval",hipError_t);
  rval = 4.0F;
  if ( i == 1 ) {
    fval = powf(rval,2.0F);
  } else {
    fval = powf(rval,2.0F);
  }

  dim3 cuda_threads1(1);
  dim3 cuda_grids1(1);

  unsigned int timer = 0;


  hipEventRecord( start, 0 );
  hipEventSynchronize(start);
  // setup execution parameters 
  // execute the kernel 
  intrinsics_Kernel1<<< cuda_grids1, cuda_threads1 >>>(D_val,D_rval);
  hipDeviceSynchronize();

  hipEventRecord( stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime( &elapsed_time_ms, start, stop );
  runtime += elapsed_time_ms;
  //  printf("intrinsics kernel execution time: %.2f ms\n",runtime);
  hipError_t = hipMemcpy(&val,D_val,(1)*sizeof(int),hipMemcpyDeviceToHost);
  cudaErrorChk("intrinsics.f90.ACC.1",28,"val",hipError_t);
  hipError_t = hipMemcpy(&rval,D_rval,(1)*sizeof(float),hipMemcpyDeviceToHost);
  cudaErrorChk("intrinsics.f90.ACC.1",28,"rval",hipError_t);
  *val__G = val;
  hipError_t = hipFree(D_val);
  cudaErrorChk("intrinsics.f90.ACC.1",29,"val",hipError_t);
  *rval__G = rval;
  hipError_t = hipFree(D_rval);
  cudaErrorChk("intrinsics.f90.ACC.1",29,"rval",hipError_t);

  return;
}
