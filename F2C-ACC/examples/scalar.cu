#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "ftocmacros.h"
#include "F2Cinclude.h"
#define TRUE 1
#define FALSE 0
typedef struct {
  int ims,ime,kms,kme;
} scalar_s1;
//!ACC$REGION(<kme-kms+1>,<ime-ims+1>,<ims,ime,kms,kme,vol:in>,<counter_gpu,counter:inout>) BEGIN
__global__ void scalar_Kernel1(scalar_s1 scalar_k1,float *vol,int *counter_gpu,int *D_counter) {
  int ims = scalar_k1.ims;
  int ime = scalar_k1.ime;
  int kms = scalar_k1.kms;
  int kme = scalar_k1.kme;
  int counter = *D_counter;

  int ipn;
  int k;
//!acc$do parallel
  ipn = blockIdx.x+ims;
//  for (ipn=ims;ipn<=ime;ipn++) {
//!acc$thread(1) begin
if (threadIdx.x == 1) {
    for (k=kms;k<=kme;k++) {
      if ( vol[FTNREF2D(k,ipn,kme-kms+1,kms,ims)] == 2.0F ) {
        counter_gpu[FTNREF1D(ipn,ims)] = counter_gpu[FTNREF1D(ipn,ims)] + 1;
      }
    }
}
//!acc$thread end
//  }
  *D_counter = counter;

  return;
}
//!ACC$REGION END
//! Scalar Values Test:
//! Designed for 1 translation and execution
//! Mark Govett       July 2011
//! NOAA/ESRL
//!
//! Unit tests for handling of scalar values thru subroutine interfaces and
//! between kernels.  The C language is pass-by value for scalars, and pass-by
//! reference for arrays.  This test insures scalars can be passed by reference
//! and updated if necessary.
//!
//! The intent attribute is used in subroutine declarations, and used to 
//! indicate if an update of the scalar is needed.  By default, scalars are
//! pass-by-value (intent IN) to avoid problems when contants are passed into
//! the subroutine.  Users must explicitly state when intent is IN or INOUT.
//!
//! To communicate scalar results between CPU and 1, intent must be
//! specified in the ACC$REGION data section.
//!
extern "C" void scalar_ (float *vol,int *counter__G,int *ims__G,int *ime__G,int *kms__G,int *kme__G) {

  static float runtime = 0.0;
  float elapsed_time_ms = 0.f;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  int counter=*counter__G;
  int ims=*ims__G;
  int ime=*ime__G;
  int kms=*kms__G;
  int kme=*kme__G;
  int ipn;
  int *counter_gpu = (int*) malloc((ime-ims+1)*sizeof(int));
  if (counter_gpu == NULL) {
    printf("ERROR: memory allocation error");
    exit(1);
  }

hipError_t hipError_t;
  float *D_vol;
  hipError_t = hipMalloc((void **) &D_vol,((kme-kms+1)*(ime-ims+1))*sizeof(float));
  cudaErrorChk("scalar.f90.ACC.1",30,"vol",hipError_t);
  int *D_counter_gpu;
  hipError_t = hipMalloc((void **) &D_counter_gpu,((ime-ims+1))*sizeof(int));
  cudaErrorChk("scalar.f90.ACC.1",30,"counter_gpu",hipError_t);
  int *D_counter;
  hipError_t = hipMalloc((void **) &D_counter,(1)*sizeof(int));
  cudaErrorChk("scalar.f90.ACC.1",30,"counter",hipError_t);
  counter = 0;
  for (ipn=ims;ipn<=ime;ipn++) {
    counter_gpu[FTNREF1D(ipn,ims)] = counter;
  }

  dim3 cuda_threads1(kme-kms+1);
  dim3 cuda_grids1(ime-ims+1);

  unsigned int timer = 0;

  scalar_s1 scalar_k1;
  scalar_k1.ims = ims;
  scalar_k1.ime = ime;
  scalar_k1.kms = kms;
  scalar_k1.kme = kme;
  hipError_t = hipMemcpy(D_vol,vol,((kme-kms+1)*(ime-ims+1))*sizeof(float),hipMemcpyHostToDevice);
  cudaErrorChk("scalar.f90.ACC.1",30,"vol",hipError_t);
  hipError_t = hipMemcpy(D_counter_gpu,counter_gpu,((ime-ims+1))*sizeof(int),hipMemcpyHostToDevice);
  cudaErrorChk("scalar.f90.ACC.1",30,"counter_gpu",hipError_t);
  hipError_t = hipMemcpy(D_counter,&counter,(1)*sizeof(int),hipMemcpyHostToDevice);
  cudaErrorChk("scalar.f90.ACC.1",30,"counter",hipError_t);
  hipError_t = hipMemcpy(D_counter,&counter,(1)*sizeof(int),hipMemcpyHostToDevice);
  cudaErrorChk("scalar.f90.ACC.1",30,"counter",hipError_t);

  hipEventRecord( start, 0 );
  hipEventSynchronize(start);
  // setup execution parameters 
  // execute the kernel 
  scalar_Kernel1<<< cuda_grids1, cuda_threads1 >>>(scalar_k1,D_vol,D_counter_gpu,D_counter);
  hipDeviceSynchronize();

  hipEventRecord( stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime( &elapsed_time_ms, start, stop );
  runtime += elapsed_time_ms;
  //  printf("scalar kernel execution time: %.2f ms\n",runtime);
  hipError_t = hipMemcpy(counter_gpu,D_counter_gpu,((ime-ims+1))*sizeof(int),hipMemcpyDeviceToHost);
  cudaErrorChk("scalar.f90.ACC.1",41,"counter_gpu",hipError_t);
  hipError_t = hipMemcpy(&counter,D_counter,(1)*sizeof(int),hipMemcpyDeviceToHost);
  cudaErrorChk("scalar.f90.ACC.1",41,"counter",hipError_t);
  for (ipn=ims;ipn<=ime;ipn++) {
    counter = counter + counter_gpu[FTNREF1D(ipn,ims)];
  }
  hipError_t = hipFree(D_vol);
  cudaErrorChk("scalar.f90.ACC.1",45,"vol",hipError_t);
  free(counter_gpu);
  hipError_t = hipFree(D_counter_gpu);
  cudaErrorChk("scalar.f90.ACC.1",45,"counter_gpu",hipError_t);
  *counter__G = counter;
  hipError_t = hipFree(D_counter);
  cudaErrorChk("scalar.f90.ACC.1",45,"counter",hipError_t);

  return;
}
