#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "ftocmacros.h"
#include "F2Cinclude.h"
#define TRUE 1
#define FALSE 0
#define NZ 50
#define NX 50
#define ITER 2
   __shared__ int operand3dS[(NZ)*(NX)*(ITER)];
typedef struct {
  int iter,nz,nx;
} shared3dout_s1;
//!ACC$REGION(<nz>,<nx>,<operand3d:out>,<iter,nz,nx:in>,<operand3d:inout,shared>) BEGIN
__global__ void shared3dout_Kernel1(int *operand3d,shared3dout_s1 shared3dout_k1) {
// Generated copy from GPU global to shared memory
  int iter = shared3dout_k1.iter;
  int nz = shared3dout_k1.nz;
  int nx = shared3dout_k1.nx;

  int i;
  int k;
  int it;
//!ACC$DO PARALLEL(1)
  i = blockIdx.x+1;
//  for (i=1;i<=nx;i++) {
//!ACC$DO VECTOR(1)
    k = threadIdx.x+1;
//    for (k=1;k<=nz;k++) {
      for (it=1;it<=iter;it++) {
        operand3dS[FTNREF3D(k,i,it,nz,nx,1,1,1)] = (i - 1) * nx + k;
      }
//    }
//  }
// Generated copy from GPU shared to global memory
  int iS1,iS2,iS3,iS4,iS5,iS6,iS7;
  for (iS3=1;iS3<=iter;iS3++) {
    operand3d[FTNREF3D(threadIdx.x+1,blockIdx.x+1,iS3,nz,nx,1,1,1)] = operand3dS[FTNREF3D(threadIdx.x,blockIdx.x+1,iS3,nz,nx,0,1,1)];
  }

  return;
}
//!ACC$REGION END
//! Unit tests to exercise copies between GPU global and shared memory
//! as needed given by the intent of the variable as specified in the 
//! ACC$REGION
//! To copy from global to shared, an intent IN must be given in the shared
//! memory variable entry (eg. <operand:in,shared> ).
//!
extern "C" void shared3dout_ (int *nx__G,int *nz__G,int *iter__G,int *operand3d) {

  static float runtime = 0.0;
  float elapsed_time_ms = 0.f;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  int nx=*nx__G;
  int nz=*nz__G;
  int iter=*iter__G;
hipError_t hipError_t;
  int *D_operand3d;
  hipError_t = hipMalloc((void **) &D_operand3d,((nz)*(nx)*(iter))*sizeof(int));
  cudaErrorChk("GlobalSharedCopy.f90.ACC.1",13,"operand3d",hipError_t);
//!ACC$DATA(<operand3d:none,shared>)


  dim3 cuda_threads1(nz);
  dim3 cuda_grids1(nx);

  unsigned int timer = 0;

  shared3dout_s1 shared3dout_k1;
  shared3dout_k1.iter = iter;
  shared3dout_k1.nz = nz;
  shared3dout_k1.nx = nx;

  hipEventRecord( start, 0 );
  hipEventSynchronize(start);
  // setup execution parameters 
  // execute the kernel 
  shared3dout_Kernel1<<< cuda_grids1, cuda_threads1 >>>(D_operand3d,shared3dout_k1);
  hipDeviceSynchronize();

  hipEventRecord( stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime( &elapsed_time_ms, start, stop );
  runtime += elapsed_time_ms;
  //  printf("shared3dout kernel execution time: %.2f ms\n",runtime);
  hipError_t = hipMemcpy(operand3d,D_operand3d,((nz)*(nx)*(iter))*sizeof(int),hipMemcpyDeviceToHost);
  cudaErrorChk("GlobalSharedCopy.f90.ACC.1",23,"operand3d",hipError_t);
  hipError_t = hipFree(D_operand3d);
  cudaErrorChk("GlobalSharedCopy.f90.ACC.1",24,"operand3d",hipError_t);

  return;
}
